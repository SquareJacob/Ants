#include "hip/hip_runtime.h"
#include <SDL.h>
#include <SDL_image.h>
#include <SDL_ttf.h>
#include <SDL_mixer.h>
#include <iostream>
#include <stdlib.h>  
#include <crtdbg.h>   //for malloc and free
#include <set>
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#define _CRTDBG_MAP_ALLOC
#ifdef _DEBUG
#define new new( _NORMAL_BLOCK, __FILE__, __LINE__)
#endif

SDL_Window* window;
SDL_Renderer* renderer;
bool running;
SDL_Event event;
std::set<std::string> keys;
std::set<std::string> currentKeys;
int mouseX = 0;
int mouseY = 0;
int mouseDeltaX = 0;
int mouseDeltaY = 0;
int mouseScroll = 0;
std::set<int> buttons;
std::set<int> currentButtons;
const int WIDTH = 800;
const int HEIGHT = 600;

void debug(int line, std::string file) {
	std::cout << "Line " << line << " in file " << file << ": " << SDL_GetError() << std::endl;
}

double random() {
	return static_cast<double>(rand()) / static_cast<double>(RAND_MAX);
}

const int BRUSHSIZE = 30;

double foodPheremones[HEIGHT * WIDTH] = { 0.0 };
double homePheremones[HEIGHT * WIDTH] = { 0.0 };
const int MAXFOODPERPIXEL = 5;
int food[HEIGHT * WIDTH] = { 0 };

double speed = 1.0;
double trailDecay = 0.01;
double strengthDecay = 0.001;
double sensorDistance = 10.0;
double sensorAngle = M_PI / 4;
double rotateAmount = M_PI / 16;
const Uint32 red = 0x01000000, green = 0x00010000, blue = 0x00000100;
class Ant {
public:
	uint8_t r = 0, g = 0, b = 0;
	bool hasFood = false;
	double x = 0.0, y = 0.0, angle = 0.0, colonyX = 0.0, colonyY = 0.0, colonyRadius = 0.0, strength = 1.0;
	void draw(Uint32* pixel_ptr) {
		pixel_ptr[static_cast<int>(y) * WIDTH + static_cast<int>(x)] = red * r + green * g + blue * b + 255;
	}
	bool move() {
		double deltaX = speed * cos(angle);
		double deltaY = speed * sin(angle);
		if (0.0 < x + deltaX && x + deltaX < WIDTH && 0.0 < y + deltaY && y + deltaY < HEIGHT) {
			x += deltaX;
			y += deltaY;
			if (strength > 0.0) {
				strength -= strengthDecay;
				if (strength < 0.0) {
					strength = 0.0;
				}
			}
			return true;
		}
		else {
			angle = random() * 2.0 * M_PI;
			return false;
		}
	}
	void trail() {
		if (hasFood) {
			foodPheremones[static_cast<int>(y) * WIDTH + static_cast<int>(x)] = strength;
		}
		else {
			homePheremones[static_cast<int>(y) * WIDTH + static_cast<int>(x)] = strength;
			if (food[static_cast<int>(y) * WIDTH + static_cast<int>(x)] > 0) {
				food[static_cast<int>(y) * WIDTH + static_cast<int>(x)]--;
				hasFood = true;
				r = 255;
				b = 0;
				angle += M_PI;
				strength = 1.0;
			}
		}
		if ((x - colonyX) * (x - colonyX) + (y - colonyY) * (y - colonyY) < colonyRadius * colonyRadius) {
			hasFood = false;
			r = 0;
			b = 255;
			angle -= M_PI;
			move();
			strength = 1.0;
		}
	}
	void sense() {
		double frontSensor;
		double leftSensor;
		double rightSensor;
		if (hasFood) {
			frontSensor = homePheremones[static_cast<int>(y + sensorDistance * sin(angle)) * WIDTH + static_cast<int>(x + sensorDistance * cos(angle))];
			leftSensor = homePheremones[static_cast<int>(y + sensorDistance * sin(angle + sensorAngle)) * WIDTH + static_cast<int>(x + sensorDistance * cos(angle + sensorAngle))];
			rightSensor = homePheremones[static_cast<int>(y + sensorDistance * sin(angle - sensorAngle)) * WIDTH + static_cast<int>(x + sensorDistance * cos(angle - sensorAngle))];
		}
		else {
			frontSensor = foodPheremones[static_cast<int>(y + sensorDistance * sin(angle)) * WIDTH + static_cast<int>(x + sensorDistance * cos(angle))];
			leftSensor = foodPheremones[static_cast<int>(y + sensorDistance * sin(angle + sensorAngle)) * WIDTH + static_cast<int>(x + sensorDistance * cos(angle + sensorAngle))];
			rightSensor = foodPheremones[static_cast<int>(y + sensorDistance * sin(angle - sensorAngle)) * WIDTH + static_cast<int>(x + sensorDistance * cos(angle - sensorAngle))];
		}
		if (frontSensor > leftSensor && frontSensor > rightSensor) {
			return;
		}
		else if (frontSensor < leftSensor && frontSensor < rightSensor) {
			angle += static_cast<float>((2 * (rand() % 2)) - 1) * rotateAmount;
		}
		else if (rightSensor > leftSensor) {
			angle -= rotateAmount;
		}
		else if (rightSensor < leftSensor) {
			angle += rotateAmount;
		}
	}
};

const int ANTS = 25000;
class Colony {
public:
	uint8_t r = 0, g = 0, b = 255;
	int radius = 15.0, x = WIDTH / 2, y = HEIGHT / 2;
	Ant ants[ANTS];
	void draw(Uint32* pixel_ptr) {
		for (int i = -radius; i <= radius; i++) {
			for (int j = -radius; j <= radius; j++) {
				if (i * i + j * j < radius * radius) {
					pixel_ptr[(y + j) * WIDTH + (x + i)] = red * r + green * g + blue * b + 255;
				}
			}
		}
	}
	void setup() {
		Ant* a;
		double angle;
		for (int i = 0; i < ANTS; i++) {
			a = &ants[i];
			a->r = r;
			a->g = g;
			a->b = b;
			angle = random() * 2.0 * M_PI;
			a->x = x + radius * cos(angle);
			a->y = y + radius * sin(angle);
			a->colonyX = x;
			a->colonyY = y;
			a->colonyRadius = radius;
			a->angle = angle;
		}
	}
};
Colony colony;

int main(int argc, char* argv[]) {
	if (SDL_Init(SDL_INIT_EVERYTHING) == 0 && TTF_Init() == 0 && Mix_OpenAudio(44100, MIX_DEFAULT_FORMAT, 2, 2048) == 0) {
		//Setup
		window = SDL_CreateWindow("Window", SDL_WINDOWPOS_CENTERED, SDL_WINDOWPOS_CENTERED, WIDTH, HEIGHT, 0);
		if (window == NULL) {
			debug(__LINE__, __FILE__);
			return 0;
		}

		renderer = SDL_CreateRenderer(window, -1, 0);
		if (renderer == NULL) {
			debug(__LINE__, __FILE__);
			return 0;
		}
		srand(time(0));
		colony.setup();

		SDL_Texture* texture = SDL_CreateTexture(renderer, SDL_PIXELFORMAT_RGBA8888,
			SDL_TEXTUREACCESS_STREAMING, WIDTH, HEIGHT);
		void* txtPixels;
		int pitch;
		SDL_PixelFormat* format = SDL_AllocFormat(SDL_PIXELFORMAT_RGBA8888);
		Uint32* pixel_ptr;

		//Main loop
		running = true;
		while (running) {
			//handle events
			for (std::string i : keys) {
				currentKeys.erase(i); //make sure only newly pressed keys are in currentKeys
			}
			for (int i : buttons) {
				currentButtons.erase(i); //make sure only newly pressed buttons are in currentButtons
			}
			mouseScroll = 0;
			while (SDL_PollEvent(&event)) {
				switch (event.type) {
				case SDL_QUIT:
					running = false;
					break;
				case SDL_KEYDOWN:
					if (!keys.contains(std::string(SDL_GetKeyName(event.key.keysym.sym)))) {
						currentKeys.insert(std::string(SDL_GetKeyName(event.key.keysym.sym)));
					}
					keys.insert(std::string(SDL_GetKeyName(event.key.keysym.sym))); //add keydown to keys set
					break;
				case SDL_KEYUP:
					keys.erase(std::string(SDL_GetKeyName(event.key.keysym.sym))); //remove keyup from keys set
					break;
				case SDL_MOUSEMOTION:
					mouseX = event.motion.x;
					mouseY = event.motion.y;
					mouseDeltaX = event.motion.xrel;
					mouseDeltaY = event.motion.yrel;
					break;
				case SDL_MOUSEBUTTONDOWN:
					if (!buttons.contains(event.button.button)) {
						currentButtons.insert(event.button.button);
					}
					buttons.insert(event.button.button);
					break;
				case SDL_MOUSEBUTTONUP:
					buttons.erase(event.button.button);
					break;
				case SDL_MOUSEWHEEL:
					mouseScroll = event.wheel.y;
					break;
				}
			}

			if (currentButtons.contains(1) || buttons.contains(3)) {
				for (int i = -BRUSHSIZE; i <= BRUSHSIZE; i++) {
					for (int j = -BRUSHSIZE; j <= BRUSHSIZE; j++) {
						if (i * i + j * j < BRUSHSIZE * BRUSHSIZE) {
							if (food[(mouseY + j) * WIDTH + (mouseX + i)] < MAXFOODPERPIXEL) {
								food[(mouseY + j) * WIDTH + (mouseX + i)]++;
							}
						}
					}
				}
			}

			for (int i = 0; i < WIDTH * HEIGHT; i++) {
				if (foodPheremones[i] > 0.0) {
					foodPheremones[i] -= trailDecay;
					if (foodPheremones[i] < 0.0) {
						foodPheremones[i] = 0.0;
					}
				}
				if (homePheremones[i] > 0.0) {
					homePheremones[i] -= trailDecay;
					if (homePheremones[i] < 0.0) {
						homePheremones[i] = 0.0;
					}
				}
			}

			Ant* a;
			for (int i = 0; i < ANTS; i++) {
				a = &colony.ants[i];
				if (a->move()) {
					a->trail();
				}
			}

			SDL_LockTexture(texture, NULL, &txtPixels, &pitch);
			pixel_ptr = (Uint32*)txtPixels;
			for (int i = 0; i < WIDTH * HEIGHT; i++) {
				pixel_ptr[i] = (food[i] * 255 / MAXFOODPERPIXEL) * green + 255;
			}
			for (int i = 0; i < ANTS; i++) {
				a = &colony.ants[i];
				a->sense();
				a->draw(pixel_ptr);
			}
			colony.draw(pixel_ptr);
			SDL_UnlockTexture(texture);
			SDL_RenderCopy(renderer, texture, NULL, NULL);
			SDL_RenderPresent(renderer);
		}

		//Clean up
		if (window) {
			SDL_DestroyWindow(window);
		}
		if (renderer) {
			SDL_DestroyRenderer(renderer);
		}
		TTF_Quit();
		Mix_Quit();
		IMG_Quit();
		SDL_Quit();
		return 0;
	}
	else {
		return 0;
	}
}