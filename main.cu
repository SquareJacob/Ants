#include "hip/hip_runtime.h"
#include <SDL.h>
#include <SDL_image.h>
#include <SDL_ttf.h>
#include <SDL_mixer.h>
#include <iostream>
#include <stdlib.h>  
#include <crtdbg.h>   //for malloc and free
#include <set>
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#define _CRTDBG_MAP_ALLOC
#ifdef _DEBUG
#define new new( _NORMAL_BLOCK, __FILE__, __LINE__)
#endif

SDL_Window* window;
SDL_Renderer* renderer;
bool running;
SDL_Event event;
std::set<std::string> keys;
std::set<std::string> currentKeys;
int mouseX = 0;
int mouseY = 0;
int mouseDeltaX = 0;
int mouseDeltaY = 0;
int mouseScroll = 0;
std::set<int> buttons;
std::set<int> currentButtons;
const int WIDTH = 800;
const int HEIGHT = 600;

void debug(int line, std::string file) {
	std::cout << "Line " << line << " in file " << file << ": " << SDL_GetError() << std::endl;
}

double random() {
	return static_cast<double>(rand()) / static_cast<double>(RAND_MAX);
}

__device__ double mod(double m, double n) {
	double result = m;
	if (result < 0) {
		while (result < 0) {
			result += n;
		}
	}
	else {
		while (result > n) {
			result -= n;
		}
	}
	return result;
}

const int BRUSHSIZE = 30;
const int MAXFOODPERPIXEL = 9999;
int food[HEIGHT * WIDTH] = { 0 };
const int WALLSIZE = 5;
int wall[HEIGHT * WIDTH] = { 0 }, *d_wall;
size_t allInts = sizeof(int) * static_cast<size_t>(WIDTH * HEIGHT);

//strength, angle
struct Pheremone {
	double strength = 0.0;
	double angle = 0.0;
};
Pheremone foodPheremones[HEIGHT * WIDTH], *d_foodPheremones;
Pheremone homePheremones[HEIGHT * WIDTH], *d_homePheremones;
size_t allPheremones = sizeof(Pheremone) * static_cast<size_t>(WIDTH * HEIGHT);

const int ANGLESAMPLES = 11;
const int LENGTHSAMPLES = 32;
double speed = 1.0;
double trailDecay = 0.001;
double strengthDecay = 0.0001;
double antDecay = 0.00;
double sensorDistance = 10.0;
double sensorAngle = M_PI / 4;
double rotateAmountMin = M_PI / 20;
double randomRotate = M_PI / 12;
const Uint32 red = 0x01000000, green = 0x00010000, blue = 0x00000100;
class Ant {
public:
	uint8_t r = 0, g = 0, b = 0;
	bool hasFood = false;
	double x = 0.0, y = 0.0, angle = 0.0, colonyX = 0.0, colonyY = 0.0, colonyRadius = 0.0, strength = 1.0;
	void setup() {
		double angle = random() * 2.0 * M_PI;
		x = colonyX + colonyRadius * cos(angle);
		y = colonyY + colonyRadius * sin(angle);
		this->angle = angle;
		strength = 0.0;
	}
	void draw(Uint32* pixel_ptr) {
		pixel_ptr[static_cast<int>(y) * WIDTH + static_cast<int>(x)] = red * r + green * g + blue * b + 255;
	}
	bool move() {
		angle += (2.0 * random() - 1.0) * randomRotate;
		double deltaX = speed * cos(angle);
		double deltaY = speed * sin(angle);
		if (0.0 < x + deltaX && x + deltaX < WIDTH && 0.0 < y + deltaY && y + deltaY < HEIGHT && wall[static_cast<int>(y + deltaY) * WIDTH + static_cast<int>(x+ deltaX)] == 0){
			x += deltaX;
			y += deltaY;
			if (strength > 0.0) {
				strength -= strengthDecay;
				if (strength < 0.0) {
					strength = 0.0;
				}
			}
			return true;
		}
		else if (wall[static_cast<int>(y) * WIDTH + static_cast<int>(x)] == 1) {
			setup();
		}
		else {
			angle += M_PI;
			//strength = 0.0;
			return false;
		}
	}
	void trail() {
		if (hasFood) {
			foodPheremones[static_cast<int>(y) * WIDTH + static_cast<int>(x)] = { strength, angle };
		}
		else {
			homePheremones[static_cast<int>(y) * WIDTH + static_cast<int>(x)] = { strength, angle };
			if (food[static_cast<int>(y) * WIDTH + static_cast<int>(x)] > 0) {
				food[static_cast<int>(y) * WIDTH + static_cast<int>(x)]--;
				hasFood = true;
				r = 255;
				b = 0;
				angle -= M_PI;
				strength = 1.0;
			}
		}
		if ((x - colonyX) * (x - colonyX) + (y - colonyY) * (y - colonyY) < colonyRadius * colonyRadius) {
			hasFood = false;
			r = 0;
			b = 255;
			angle += M_PI;
			strength = 1.0;
		}
	}
	__device__ void sense(int* wall, Pheremone* foodPheremones, Pheremone* homePheremones, double sensorAngle, double sensorDistance, double rotateAmountMin, double antDecay) {
		Pheremone* toUse;
		Pheremone sensors[ANGLESAMPLES];
		double lengths[ANGLESAMPLES];
		int indices[ANGLESAMPLES];
		if (hasFood) {
			toUse = homePheremones;
		}
		else {
			toUse = foodPheremones;
		}
		Pheremone current;
		double length, angle1;
		int x1, y1;
		for (int i = 0; i <	ANGLESAMPLES; i++) {
			angle1 = angle + sensorAngle * (2.0 * static_cast<float>(i) / static_cast<float>(ANGLESAMPLES - 1) - 1.0);
			for (int j = 0; j < LENGTHSAMPLES; j++) {
				length = static_cast<float>(j + 1) / static_cast<float>(LENGTHSAMPLES) * sensorDistance;
				x1 = static_cast<int>(x + length * cos(angle1));
				y1 = static_cast<int>(y + length * sin(angle1));
				if (wall[y1 * WIDTH + x1] == 1 || x1 < 0 || WIDTH < x1 || y1 < 0 || HEIGHT < y1) {
					break;
				}
				current = toUse[y1 * WIDTH + x1];
				if (current.strength > sensors[i].strength) {
					sensors[i].angle = current.angle;
					sensors[i].strength = current.strength;
					lengths[i] = static_cast<float>(j + 1) / static_cast<float>(LENGTHSAMPLES);
					indices[i] = y1 * WIDTH + x1;
				}
			}
		}
		double maxStrength = sensors[0].strength;
		for (int i = 1; i < ANGLESAMPLES; i++) {
			if (sensors[i].strength > maxStrength) {
				maxStrength = sensors[i].strength;
			}
		}

		if (maxStrength > 0.0) {
			double newAngle;
			double newLength;
			for (int i = 0; i < ANGLESAMPLES; i++) {
				if (sensors[i].strength == maxStrength) {
					newAngle = sensors[i].angle;
					newLength = lengths[i];
					toUse[indices[i]].strength -= antDecay;
					break;
				}
			}
			double rotateAmount = rotateAmountMin / newLength;
			angle = mod(angle, 2.0 * M_PI);
			newAngle = mod(newAngle + M_PI, 2.0 * M_PI);
			if (abs(angle - newAngle) < rotateAmount) {
				angle = newAngle;
			}
			else if (newAngle < angle) {
				if (angle - newAngle < M_PI) {
					angle -= rotateAmount;
				}
				else {
					angle += rotateAmount;
				}
			}
			else {
				if (newAngle - angle < M_PI) {
					angle += rotateAmount;
				}
				else {
					angle -= rotateAmount;
				}
			}
		}
	}
};

const int SQRTANTS = 159;
const int ANTS = SQRTANTS * SQRTANTS;
class Colony {
public:
	uint8_t r = 0, g = 0, b = 255;
	int radius = 15.0, x = WIDTH / 2, y = HEIGHT / 2;
	Ant ants[ANTS];
	void draw(Uint32* pixel_ptr) {
		for (int i = -radius; i <= radius; i++) {
			for (int j = -radius; j <= radius; j++) {
				if (i * i + j * j < radius * radius) {
					pixel_ptr[(y + j) * WIDTH + (x + i)] = red * r + green * g + blue * b + 255;
				}
			}
		}
	}
	void setup() {
		Ant* a;
		for (int i = 0; i < ANTS; i++) {
			a = &ants[i];
			a->r = r;
			a->g = g;
			a->b = b;
			a->colonyX = x;
			a->colonyY = y;
			a->colonyRadius = radius;
			a->setup();
		}
	}
};
Colony colony, * d_colony;
size_t s_colony = sizeof(colony);

__global__ void sense(Colony* colony, int* wall, Pheremone* foodPheremones, Pheremone* homePheremones, double sensorAngle, double sensorDistance, double rotateAmountMin, double antDecay) {
	colony->ants[threadIdx.x * SQRTANTS + blockIdx.x].sense(wall, foodPheremones, homePheremones, sensorAngle, sensorDistance, rotateAmountMin, antDecay);
}

int main(int argc, char* argv[]) {
	if (SDL_Init(SDL_INIT_EVERYTHING) == 0 && TTF_Init() == 0 && Mix_OpenAudio(44100, MIX_DEFAULT_FORMAT, 2, 2048) == 0) {
		//Setup
		window = SDL_CreateWindow("Window", SDL_WINDOWPOS_CENTERED, SDL_WINDOWPOS_CENTERED, WIDTH, HEIGHT, 0);
		if (window == NULL) {
			debug(__LINE__, __FILE__);
			return 0;
		}

		renderer = SDL_CreateRenderer(window, -1, 0);
		if (renderer == NULL) {
			debug(__LINE__, __FILE__);
			return 0;
		}
		srand(time(0));
		colony.setup();
		for (int i = 0; i < WIDTH * HEIGHT; i++) {
			food[i] = 0;
		}

		hipSetDevice(0);
		hipMalloc((void**)&d_colony, s_colony);
		hipMalloc((void**)&d_wall, allInts);
		hipMalloc((void**)&d_foodPheremones, allPheremones);
		hipMalloc((void**)&d_homePheremones, allPheremones);

		SDL_Texture* texture = SDL_CreateTexture(renderer, SDL_PIXELFORMAT_RGBA8888,
			SDL_TEXTUREACCESS_STREAMING, WIDTH, HEIGHT);
		void* txtPixels;
		int pitch;
		SDL_PixelFormat* format = SDL_AllocFormat(SDL_PIXELFORMAT_RGBA8888);
		Uint32* pixel_ptr;

		//Main loop
		running = true;
		bool playing = true;
		while (running) {
			//handle events
			for (std::string i : keys) {
				currentKeys.erase(i); //make sure only newly pressed keys are in currentKeys
			}
			for (int i : buttons) {
				currentButtons.erase(i); //make sure only newly pressed buttons are in currentButtons
			}
			mouseScroll = 0;
			while (SDL_PollEvent(&event)) {
				switch (event.type) {
				case SDL_QUIT:
					running = false;
					break;
				case SDL_KEYDOWN:
					if (!keys.contains(std::string(SDL_GetKeyName(event.key.keysym.sym)))) {
						currentKeys.insert(std::string(SDL_GetKeyName(event.key.keysym.sym)));
					}
					keys.insert(std::string(SDL_GetKeyName(event.key.keysym.sym))); //add keydown to keys set
					break;
				case SDL_KEYUP:
					keys.erase(std::string(SDL_GetKeyName(event.key.keysym.sym))); //remove keyup from keys set
					break;
				case SDL_MOUSEMOTION:
					mouseX = event.motion.x;
					mouseY = event.motion.y;
					mouseDeltaX = event.motion.xrel;
					mouseDeltaY = event.motion.yrel;
					break;
				case SDL_MOUSEBUTTONDOWN:
					if (!buttons.contains(event.button.button)) {
						currentButtons.insert(event.button.button);
					}
					buttons.insert(event.button.button);
					break;
				case SDL_MOUSEBUTTONUP:
					buttons.erase(event.button.button);
					break;
				case SDL_MOUSEWHEEL:
					mouseScroll = event.wheel.y;
					break;
				}
			}

			if (buttons.contains(1)) {
				for (int i = -BRUSHSIZE; i <= BRUSHSIZE; i++) {
					for (int j = -BRUSHSIZE; j <= BRUSHSIZE; j++) {
						if (i * i + j * j < BRUSHSIZE * BRUSHSIZE && 0 <= mouseX + i && mouseX + i <= WIDTH && 0 <= mouseY + j && mouseY + j <= HEIGHT) {
							food[(mouseY + j) * WIDTH + (mouseX + i)] = MAXFOODPERPIXEL;
						}
					}
				}
			}

			if (buttons.contains(3)) {
				for (int i = -WALLSIZE; i <= WALLSIZE; i++) {
					for (int j = -WALLSIZE; j <= WALLSIZE; j++) {
						if (i * i + j * j < WALLSIZE * WALLSIZE && 0 <= mouseX + i && mouseX + i <= WIDTH && 0 <= mouseY + j && mouseY + j <= HEIGHT) {
							wall[(mouseY + j) * WIDTH + (mouseX + i)] = 1;
						}
					}
				}
			}
			else if (buttons.contains(2)) {
				for (int i = -BRUSHSIZE; i <= BRUSHSIZE; i++) {
					for (int j = -BRUSHSIZE; j <= BRUSHSIZE; j++) {
						if (i * i + j * j < BRUSHSIZE * BRUSHSIZE && 0 <= mouseX + i && mouseX + i <= WIDTH && 0 <= mouseY + j && mouseY + j <= HEIGHT) {
							wall[(mouseY + j) * WIDTH + (mouseX + i)] = 0;
							food[(mouseY + j) * WIDTH + (mouseX + i)] = 0;
						}
					}
				}
			}

			if (currentKeys.contains("Space")) {
				playing = !playing;
			}

			Ant* a;
			if (playing) {
				for (int i = 0; i < WIDTH * HEIGHT; i++) {
					if (foodPheremones[i].strength > 0.0) {
						foodPheremones[i].strength -= trailDecay;
						if (foodPheremones[i].strength < 0.0) {
							foodPheremones[i].strength = 0.0;
						}
					}
					if (homePheremones[i].strength > 0.0) {
						homePheremones[i].strength -= trailDecay;
						if (homePheremones[i].strength < 0.0) {
							homePheremones[i].strength = 0.0;
						}
					}
				}

				for (int i = 0; i < ANTS; i++) {
					a = &colony.ants[i];
					if (a->move()) {
						a->trail();
					}
				}

				hipMemcpy(d_colony, &colony, s_colony, hipMemcpyHostToDevice);
				hipMemcpy(d_wall, wall, allInts, hipMemcpyHostToDevice);
				hipMemcpy(d_foodPheremones, foodPheremones, allPheremones, hipMemcpyHostToDevice);
				hipMemcpy(d_homePheremones, homePheremones, allPheremones, hipMemcpyHostToDevice);
				sense << <SQRTANTS, SQRTANTS >> > (d_colony, d_wall, d_foodPheremones, d_homePheremones, sensorAngle, sensorDistance, rotateAmountMin, antDecay);
				hipDeviceSynchronize();
				hipMemcpy(&colony, d_colony, s_colony, hipMemcpyDeviceToHost);
				hipMemcpy(wall, d_wall, allInts, hipMemcpyDeviceToHost);
				hipMemcpy(foodPheremones, d_foodPheremones, allPheremones, hipMemcpyDeviceToHost);
				hipMemcpy(homePheremones, d_homePheremones, allPheremones, hipMemcpyDeviceToHost);
			}

			SDL_LockTexture(texture, NULL, &txtPixels, &pitch);
			pixel_ptr = (Uint32*)txtPixels;
			for (int i = 0; i < WIDTH * HEIGHT; i++) {
				if (wall[i] == 0) {
					pixel_ptr[i] = (food[i] * 255 / MAXFOODPERPIXEL) * green + 255;
				}
				else {
					pixel_ptr[i] = (red + green + blue) * 127 + 255;
				}
			}
			for (int i = 0; i < ANTS; i++) {
				colony.ants[i].draw(pixel_ptr);
			}
			colony.draw(pixel_ptr);
			SDL_UnlockTexture(texture);
			SDL_RenderCopy(renderer, texture, NULL, NULL);
			SDL_RenderPresent(renderer);
		}

		//Clean up
		hipFree(d_colony);
		hipFree(d_wall);
		hipFree(d_foodPheremones);
		hipFree(d_homePheremones);
		if (window) {
			SDL_DestroyWindow(window);
		}
		if (renderer) {
			SDL_DestroyRenderer(renderer);
		}
		TTF_Quit();
		Mix_Quit();
		IMG_Quit();
		SDL_Quit();
		return 0;
	}
	else {
		return 0;
	}
}