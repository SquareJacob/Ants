#include "hip/hip_runtime.h"
#include <SDL.h>
#include <SDL_image.h>
#include <SDL_ttf.h>
#include <SDL_mixer.h>
#include <iostream>
#include <stdlib.h>  
#include <crtdbg.h>   //for malloc and free
#include <set>
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#define _CRTDBG_MAP_ALLOC
#ifdef _DEBUG
#define new new( _NORMAL_BLOCK, __FILE__, __LINE__)
#endif

SDL_Window* window;
SDL_Renderer* renderer;
bool running;
SDL_Event event;
std::set<std::string> keys;
std::set<std::string> currentKeys;
int mouseX = 0;
int mouseY = 0;
int mouseDeltaX = 0;
int mouseDeltaY = 0;
int mouseScroll = 0;
std::set<int> buttons;
std::set<int> currentButtons;
const int WIDTH = 800;
const int HEIGHT = 600;

void debug(int line, std::string file) {
	std::cout << "Line " << line << " in file " << file << ": " << SDL_GetError() << std::endl;
}

double random() {
	return static_cast<double>(rand()) / static_cast<double>(RAND_MAX);
}

double mod(double m, double n) {
	double result = m;
	if (result < 0) {
		while (result < 0) {
			result += n;
		}
	}
	else {
		while (result > n) {
			result -= n;
		}
	}
	return result;
}

const int BRUSHSIZE = 30;
const int MAXFOODPERPIXEL = 5;
int food[HEIGHT * WIDTH] = { 0 };

//strength, angle
struct Pheremone {
	double strength = 0.0;
	double angle = 0.0;
};
Pheremone foodPheremones[HEIGHT * WIDTH];
Pheremone homePheremones[HEIGHT * WIDTH];

double speed = 1.0;
double trailDecay = 0.01;
double strengthDecay = 0.001;
double sensorDistance = 0.0;
double sensorAngle = M_PI / 4;
double rotateAmount = M_PI / 6;
double randomRotate = M_PI / 12;
const Uint32 red = 0x01000000, green = 0x00010000, blue = 0x00000100;
class Ant {
public:
	uint8_t r = 0, g = 0, b = 0;
	bool hasFood = false;
	double x = 0.0, y = 0.0, angle = 0.0, colonyX = 0.0, colonyY = 0.0, colonyRadius = 0.0, strength = 1.0;
	void draw(Uint32* pixel_ptr) {
		pixel_ptr[static_cast<int>(y) * WIDTH + static_cast<int>(x)] = red * r + green * g + blue * b + 255;
	}
	bool move() {
		angle += (2.0 * random() - 1.0) * randomRotate;
		double deltaX = speed * cos(angle);
		double deltaY = speed * sin(angle);
		if (0.0 < x + deltaX && x + deltaX < WIDTH && 0.0 < y + deltaY && y + deltaY < HEIGHT) {
			x += deltaX;
			y += deltaY;
			if (strength > 0.0) {
				strength -= strengthDecay;
				if (strength < 0.0) {
					strength = 0.0;
				}
			}
			return true;
		}
		else {
			angle = random() * 2.0 * M_PI;
			strength = 0.0;
			return false;
		}
	}
	void trail() {
		if (hasFood) {
			if (foodPheremones[static_cast<int>(y) * WIDTH + static_cast<int>(x)].strength < strength) {
				foodPheremones[static_cast<int>(y) * WIDTH + static_cast<int>(x)] = { strength, angle };
			}
		}
		else {
			if (homePheremones[static_cast<int>(y) * WIDTH + static_cast<int>(x)].strength < strength) {
				homePheremones[static_cast<int>(y) * WIDTH + static_cast<int>(x)] = { strength, angle };
			}
			if (food[static_cast<int>(y) * WIDTH + static_cast<int>(x)] > 0) {
				food[static_cast<int>(y) * WIDTH + static_cast<int>(x)]--;
				hasFood = true;
				r = 255;
				b = 0;
				angle -= M_PI;
				strength = 1.0;
			}
		}
		if ((x - colonyX) * (x - colonyX) + (y - colonyY) * (y - colonyY) < colonyRadius * colonyRadius) {
			hasFood = false;
			r = 0;
			b = 255;
			angle += M_PI;
			strength = 1.0;
		}
	}
	void sense() {
		Pheremone frontSensor;
		Pheremone leftSensor;
		Pheremone rightSensor;
		if (hasFood) {
			frontSensor = homePheremones[static_cast<int>(y + sensorDistance * sin(angle)) * WIDTH + static_cast<int>(x + sensorDistance * cos(angle))];
			leftSensor = homePheremones[static_cast<int>(y + sensorDistance * sin(angle + sensorAngle)) * WIDTH + static_cast<int>(x + sensorDistance * cos(angle + sensorAngle))];
			rightSensor = homePheremones[static_cast<int>(y + sensorDistance * sin(angle - sensorAngle)) * WIDTH + static_cast<int>(x + sensorDistance * cos(angle - sensorAngle))];
		}
		else {
			frontSensor = foodPheremones[static_cast<int>(y + sensorDistance * sin(angle)) * WIDTH + static_cast<int>(x + sensorDistance * cos(angle))];
			leftSensor = foodPheremones[static_cast<int>(y + sensorDistance * sin(angle + sensorAngle)) * WIDTH + static_cast<int>(x + sensorDistance * cos(angle + sensorAngle))];
			rightSensor = foodPheremones[static_cast<int>(y + sensorDistance * sin(angle - sensorAngle)) * WIDTH + static_cast<int>(x + sensorDistance * cos(angle - sensorAngle))];
		}
		double maxStrength = std::max(frontSensor.strength, std::max(leftSensor.strength, rightSensor.strength));
		if (maxStrength > 0.0) {
			double newAngle;
			if (frontSensor.strength == maxStrength) {
				newAngle = frontSensor.angle;
			}
			else if (leftSensor.strength == maxStrength) {
				newAngle = leftSensor.angle;
			}
			else if (rightSensor.strength == maxStrength) {
				newAngle = rightSensor.angle;
			}
			//angle = newAngle + M_PI;
			angle = mod(angle, 2.0 * M_PI);
			newAngle = mod(newAngle + M_PI, 2.0 * M_PI);
			if (abs(angle - newAngle) < rotateAmount) {
				angle = newAngle;
			}
			else if (newAngle < angle) {
				if (angle - newAngle < M_PI) {
					angle -= rotateAmount;
				}
				else {
					angle += rotateAmount;
				}
			}
			else {
				if (newAngle - angle < M_PI) {
					angle += rotateAmount;
				}
				else {
					angle -= rotateAmount;
				}
			}
		}
	}
};

const int ANTS = 25000;
class Colony {
public:
	uint8_t r = 0, g = 0, b = 255;
	int radius = 15.0, x = WIDTH / 2, y = HEIGHT / 2;
	Ant ants[ANTS];
	void draw(Uint32* pixel_ptr) {
		for (int i = -radius; i <= radius; i++) {
			for (int j = -radius; j <= radius; j++) {
				if (i * i + j * j < radius * radius) {
					pixel_ptr[(y + j) * WIDTH + (x + i)] = red * r + green * g + blue * b + 255;
				}
			}
		}
	}
	void setup() {
		Ant* a;
		double angle;
		for (int i = 0; i < ANTS; i++) {
			a = &ants[i];
			a->r = r;
			a->g = g;
			a->b = b;
			angle = random() * 2.0 * M_PI;
			a->x = x + radius * cos(angle);
			a->y = y + radius * sin(angle);
			a->colonyX = x;
			a->colonyY = y;
			a->colonyRadius = radius;
			a->angle = angle;
		}
	}
};
Colony colony;

int main(int argc, char* argv[]) {
	if (SDL_Init(SDL_INIT_EVERYTHING) == 0 && TTF_Init() == 0 && Mix_OpenAudio(44100, MIX_DEFAULT_FORMAT, 2, 2048) == 0) {
		//Setup
		window = SDL_CreateWindow("Window", SDL_WINDOWPOS_CENTERED, SDL_WINDOWPOS_CENTERED, WIDTH, HEIGHT, 0);
		if (window == NULL) {
			debug(__LINE__, __FILE__);
			return 0;
		}

		renderer = SDL_CreateRenderer(window, -1, 0);
		if (renderer == NULL) {
			debug(__LINE__, __FILE__);
			return 0;
		}
		srand(time(0));
		colony.setup();

		SDL_Texture* texture = SDL_CreateTexture(renderer, SDL_PIXELFORMAT_RGBA8888,
			SDL_TEXTUREACCESS_STREAMING, WIDTH, HEIGHT);
		void* txtPixels;
		int pitch;
		SDL_PixelFormat* format = SDL_AllocFormat(SDL_PIXELFORMAT_RGBA8888);
		Uint32* pixel_ptr;

		//Main loop
		running = true;
		while (running) {
			//handle events
			for (std::string i : keys) {
				currentKeys.erase(i); //make sure only newly pressed keys are in currentKeys
			}
			for (int i : buttons) {
				currentButtons.erase(i); //make sure only newly pressed buttons are in currentButtons
			}
			mouseScroll = 0;
			while (SDL_PollEvent(&event)) {
				switch (event.type) {
				case SDL_QUIT:
					running = false;
					break;
				case SDL_KEYDOWN:
					if (!keys.contains(std::string(SDL_GetKeyName(event.key.keysym.sym)))) {
						currentKeys.insert(std::string(SDL_GetKeyName(event.key.keysym.sym)));
					}
					keys.insert(std::string(SDL_GetKeyName(event.key.keysym.sym))); //add keydown to keys set
					break;
				case SDL_KEYUP:
					keys.erase(std::string(SDL_GetKeyName(event.key.keysym.sym))); //remove keyup from keys set
					break;
				case SDL_MOUSEMOTION:
					mouseX = event.motion.x;
					mouseY = event.motion.y;
					mouseDeltaX = event.motion.xrel;
					mouseDeltaY = event.motion.yrel;
					break;
				case SDL_MOUSEBUTTONDOWN:
					if (!buttons.contains(event.button.button)) {
						currentButtons.insert(event.button.button);
					}
					buttons.insert(event.button.button);
					break;
				case SDL_MOUSEBUTTONUP:
					buttons.erase(event.button.button);
					break;
				case SDL_MOUSEWHEEL:
					mouseScroll = event.wheel.y;
					break;
				}
			}

			if (currentButtons.contains(1) || buttons.contains(3)) {
				for (int i = -BRUSHSIZE; i <= BRUSHSIZE; i++) {
					for (int j = -BRUSHSIZE; j <= BRUSHSIZE; j++) {
						if (i * i + j * j < BRUSHSIZE * BRUSHSIZE) {
							if (food[(mouseY + j) * WIDTH + (mouseX + i)] < MAXFOODPERPIXEL) {
								food[(mouseY + j) * WIDTH + (mouseX + i)]++;
							}
						}
					}
				}
			}

			for (int i = 0; i < WIDTH * HEIGHT; i++) {
				if (foodPheremones[i].strength > 0.0) {
					foodPheremones[i].strength -= trailDecay;
					if (foodPheremones[i].strength < 0.0) {
						foodPheremones[i].strength = 0.0;
					}
				}
				if (homePheremones[i].strength > 0.0) {
					homePheremones[i].strength -= trailDecay;
					if (homePheremones[i].strength < 0.0) {
						homePheremones[i].strength = 0.0;
					}
				}
			}

			Ant* a;
			for (int i = 0; i < ANTS; i++) {
				a = &colony.ants[i];
				if (a->move()) {
					a->trail();
				}
			}

			SDL_LockTexture(texture, NULL, &txtPixels, &pitch);
			pixel_ptr = (Uint32*)txtPixels;
			for (int i = 0; i < WIDTH * HEIGHT; i++) {
				pixel_ptr[i] = (food[i] * 255 / MAXFOODPERPIXEL) * green + 255;
			}
			for (int i = 0; i < ANTS; i++) {
				a = &colony.ants[i];
				a->sense();
				a->draw(pixel_ptr);
			}
			colony.draw(pixel_ptr);
			SDL_UnlockTexture(texture);
			SDL_RenderCopy(renderer, texture, NULL, NULL);
			SDL_RenderPresent(renderer);
		}

		//Clean up
		if (window) {
			SDL_DestroyWindow(window);
		}
		if (renderer) {
			SDL_DestroyRenderer(renderer);
		}
		TTF_Quit();
		Mix_Quit();
		IMG_Quit();
		SDL_Quit();
		return 0;
	}
	else {
		return 0;
	}
}