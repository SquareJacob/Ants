#include <SDL.h>
#include <SDL_image.h>
#include <SDL_ttf.h>
#include <SDL_mixer.h>
#include <iostream>
#include <stdlib.h>  
#include <crtdbg.h>   //for malloc and free
#include <set>
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#define _CRTDBG_MAP_ALLOC
#ifdef _DEBUG
#define new new( _NORMAL_BLOCK, __FILE__, __LINE__)
#endif

SDL_Window* window;
SDL_Renderer* renderer;
bool running;
SDL_Event event;
std::set<std::string> keys;
std::set<std::string> currentKeys;
int mouseX = 0;
int mouseY = 0;
int mouseDeltaX = 0;
int mouseDeltaY = 0;
int mouseScroll = 0;
std::set<int> buttons;
std::set<int> currentButtons;

void debug(int line, std::string file) {
	std::cout << "Line " << line << " in file " << file << ": " << SDL_GetError() << std::endl;
}

int main(int argc, char* argv[]) {
	if (SDL_Init(SDL_INIT_EVERYTHING) == 0 && TTF_Init() == 0 && Mix_OpenAudio(44100, MIX_DEFAULT_FORMAT, 2, 2048) == 0) {
		//Setup
		window = SDL_CreateWindow("Window", SDL_WINDOWPOS_CENTERED, SDL_WINDOWPOS_CENTERED, 800, 600, 0);
		if (window == NULL) {
			debug(__LINE__, __FILE__);
			return 0;
		}

		renderer = SDL_CreateRenderer(window, -1, 0);
		if (renderer == NULL) {
			debug(__LINE__, __FILE__);
			return 0;
		}

		//Main loop
		running = true;
		while (running) {
			//handle events
			for (std::string i : keys) {
				currentKeys.erase(i); //make sure only newly pressed keys are in currentKeys
			}
			for (int i : buttons) {
				currentButtons.erase(i); //make sure only newly pressed buttons are in currentButtons
			}
			mouseScroll = 0;
			while (SDL_PollEvent(&event)) {
				switch (event.type) {
				case SDL_QUIT:
					running = false;
					break;
				case SDL_KEYDOWN:
					if (!keys.contains(std::string(SDL_GetKeyName(event.key.keysym.sym)))) {
						currentKeys.insert(std::string(SDL_GetKeyName(event.key.keysym.sym)));
					}
					keys.insert(std::string(SDL_GetKeyName(event.key.keysym.sym))); //add keydown to keys set
					break;
				case SDL_KEYUP:
					keys.erase(std::string(SDL_GetKeyName(event.key.keysym.sym))); //remove keyup from keys set
					break;
				case SDL_MOUSEMOTION:
					mouseX = event.motion.x;
					mouseY = event.motion.y;
					mouseDeltaX = event.motion.xrel;
					mouseDeltaY = event.motion.yrel;
					break;
				case SDL_MOUSEBUTTONDOWN:
					if (!buttons.contains(event.button.button)) {
						currentButtons.insert(event.button.button);
					}
					buttons.insert(event.button.button);
					break;
				case SDL_MOUSEBUTTONUP:
					buttons.erase(event.button.button);
					break;
				case SDL_MOUSEWHEEL:
					mouseScroll = event.wheel.y;
					break;
				}
			}
		}

		//Clean up
		if (window) {
			SDL_DestroyWindow(window);
		}
		if (renderer) {
			SDL_DestroyRenderer(renderer);
		}
		TTF_Quit();
		Mix_Quit();
		IMG_Quit();
		SDL_Quit();
		return 0;
	}
	else {
		return 0;
	}
}